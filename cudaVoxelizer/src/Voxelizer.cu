#include "hip/hip_runtime.h"
#include "Mesh.h"
#include "hip/hip_vector_types.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void voxelize(float* faces, int n_faces, size_t dim_x, size_t dim_y, size_t dim_z, unsigned int* v_table, float3 spacing, float3 bbox_min) {
    size_t thread_idx = threadIdx.x + (blockDim.x * blockIdx.x);

    float3 delta_p = make_float3(spacing.x, spacing.y, spacing.z);
    int3 grid_size = make_int3(dim_x - 1, dim_y - 1, dim_z - 1);

    if (thread_idx < n_faces) {
        // We have 1 thread x triangle and each triangle is composed by 3 vertices.
        // Each vertex is composed by 3 float so the first vertex will be at thread_idx * 9
        size_t i = thread_idx * 9;

        // TRIANGLE INFORMATION
        float3 v0 = make_float3(faces[i], faces[i + 1], faces[i + 2]) - bbox_min;
        float3 v1 = make_float3(faces[i + 3], faces[i + 4], faces[i + 5]) - bbox_min;
        float3 v2 = make_float3(faces[i + 6], faces[i + 7], faces[i + 8]) - bbox_min;

        float3 e0 = v1 - v0;
        float3 e1 = v2 - v1;
        float3 e2 = v0 - v2;

        float3 n = normalize(cross(e0, e1));

        // Compute current triangle bbox
        Mesh::AABBox<float3> t_bbox{ fminf(v0, fminf(v1, v2)), fmaxf(v0, fmaxf(v1, v2)) };

        // Compute current triangle bbox in voxel grid coordinates
        Mesh::AABBox<int3> t_bbox_grid{};
        t_bbox_grid.p_min = clamp(make_int3(t_bbox.p_min / spacing), make_int3(0, 0, 0), grid_size);
        t_bbox_grid.p_max = clamp(make_int3(t_bbox.p_max / spacing), make_int3(0, 0, 0), grid_size);

        // SETUP STAGE
        // Compute critical point
        float3 c = make_float3(.0f, .0f, .0f);
        if (n.x > .0f) { c.x = delta_p.x; }
        if (n.y > .0f) { c.y = delta_p.y; }
        if (n.z > .0f) { c.z = delta_p.z; }

        // Compute d1 and d2 for plane overlap test
        float d1 = dot(n, c - v0);
        float d2 = dot(n, (delta_p - c) - v0);

        // Loop unrolling to prepare projection test properties
        // XY plane
        float2 e0_xy = make_float2(-1.f * e0.y, e0.x);
        float2 e1_xy = make_float2(-1.f * e1.y, e1.x);
        float2 e2_xy = make_float2(-1.f * e2.y, e2.x);
        if (n.z < .0f) {
            e0_xy = -e0_xy;
            e1_xy = -e1_xy;
            e2_xy = -e2_xy;
        }

        float d_e0_xy = ( -1.0f * dot(e0_xy, make_float2(v0.x, v0.y)) ) + fmaxf(.0f, delta_p.x * e0_xy.x) + fmaxf(.0f, delta_p.y * e0_xy.y);
        float d_e1_xy = ( -1.0f * dot(e1_xy, make_float2(v1.x, v1.y)) ) + fmaxf(.0f, delta_p.x * e1_xy.x) + fmaxf(.0f, delta_p.y * e1_xy.y);
        float d_e2_xy = ( -1.0f * dot(e2_xy, make_float2(v2.x, v2.y)) ) + fmaxf(.0f, delta_p.x * e2_xy.x) + fmaxf(.0f, delta_p.y * e2_xy.y);

        // YZ plane
        float2 e0_yz = make_float2(-1.f * e0.z, e0.y);
        float2 e1_yz = make_float2(-1.f * e1.z, e1.y);
        float2 e2_yz = make_float2(-1.f * e2.z, e2.y);
        if (n.x < .0f) {
            e0_yz = -e0_yz;
            e1_yz = -e1_yz;
            e2_yz = -e2_yz;
        }

        float d_e0_yz = ( -1.0f * dot(e0_yz, make_float2(v0.y, v0.z)) ) + fmaxf(.0f, delta_p.y * e0_yz.x) + fmaxf(.0f, delta_p.z * e0_yz.y);
        float d_e1_yz = ( -1.0f * dot(e1_yz, make_float2(v1.y, v1.z)) ) + fmaxf(.0f, delta_p.y * e1_yz.x) + fmaxf(.0f, delta_p.z * e1_yz.y);
        float d_e2_yz = ( -1.0f * dot(e2_yz, make_float2(v2.y, v2.z)) ) + fmaxf(.0f, delta_p.y * e2_yz.x) + fmaxf(.0f, delta_p.z * e2_yz.y);

        // ZX plane
        float2 e0_zx = make_float2(-1.f * e0.x, e0.z);
        float2 e1_zx = make_float2(-1.f * e1.x, e1.z);
        float2 e2_zx = make_float2(-1.f * e2.x, e2.z);
        if (n.y < .0f) {
            e0_zx = -e0_zx;
            e1_zx = -e1_zx;
            e2_zx = -e2_zx;
        }

        float d_e0_zx = ( -1.0f * dot(e0_zx, make_float2(v0.z, v0.x)) ) + fmaxf(.0f, delta_p.x * e0_zx.x) + fmaxf(.0f, delta_p.z * e0_zx.y);
        float d_e1_zx = ( -1.0f * dot(e1_zx, make_float2(v1.z, v1.x)) ) + fmaxf(.0f, delta_p.x * e1_zx.x) + fmaxf(.0f, delta_p.z * e1_zx.y);
        float d_e2_zx = ( -1.0f * dot(e2_zx, make_float2(v2.z, v2.x)) ) + fmaxf(.0f, delta_p.x * e2_zx.x) + fmaxf(.0f, delta_p.z * e2_zx.y);

        // OVERLAP TEST
        // For each voxel in the triangle bbox
        for (int z = t_bbox_grid.p_min.z; z <= t_bbox_grid.p_max.z; ++z) {
            for (int y = t_bbox_grid.p_min.y; y <= t_bbox_grid.p_max.y; ++y) {
                for (int x = t_bbox_grid.p_min.x; x <= t_bbox_grid.p_max.x; ++x) {
                    // Compute minimum corner coordinates
                    float3 p = make_float3(x * spacing.x, y * spacing.y, z * spacing.z);

                    // Triangle plane overlap test
                    if (((dot(n, p) + d1) * (dot(n, p) + d2)) > .0f) { continue; }

                    // Check if the current triangle overlaps the current voxel
                    // XY plane
                    float2 p_xy = make_float2(p.x, p.y);
                    if (dot(e0_xy, p_xy) + d_e0_xy < .0f) { continue; }
                    if (dot(e1_xy, p_xy) + d_e1_xy < .0f) { continue; }
                    if (dot(e2_xy, p_xy) + d_e2_xy < .0f) { continue; }

                    // YZ plane
                    float2 p_yz = make_float2(p.y, p.z);
                    if (dot(e0_yz, p_yz) + d_e0_yz < .0f) { continue; }
                    if (dot(e1_yz, p_yz) + d_e1_yz < .0f) { continue; }
                    if (dot(e2_yz, p_yz) + d_e2_yz < .0f) { continue; }

                    // ZX plane
                    float2 p_zx = make_float2(p.z, p.x);
                    if (dot(e0_zx, p_zx) + d_e0_zx < .0f) { continue; }
                    if (dot(e1_zx, p_zx) + d_e1_zx < .0f) { continue; }
                    if (dot(e2_zx, p_zx) + d_e2_zx < .0f) { continue; }

                    // Set the current voxel as intersected
                    size_t location = (size_t)x + ((size_t)y * dim_x) + ((size_t)z * dim_y * dim_x);
                    atomicAdd(&v_table[location], 1);

                    continue;
                }
            }
        }
    }
}

void kernelWrapper(Mesh::Mesh& m, const Mesh::VoxelGrid& v_grid, unsigned int* v_table, float& time) {
    unsigned int* v_table_d = nullptr;
    float* faces_d = nullptr;

    hipEvent_t start;
    hipEvent_t end;
    float elapsed_time;

    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&end));

    // Setup voxel table inside GPU
    gpuErrchk(hipMalloc((void**)&v_table_d, sizeof(unsigned int) * (size_t)v_grid.dim_x * (size_t)v_grid.dim_y * (size_t)v_grid.dim_z));
    gpuErrchk(hipMemcpy(v_table_d, v_table, sizeof(unsigned int) * (size_t)v_grid.dim_x * (size_t)v_grid.dim_y * (size_t)v_grid.dim_z, hipMemcpyHostToDevice));

    float* faces = (float*)calloc(m.faces_idx.size() * 9, sizeof(float));
    for (size_t i = 0; i < m.faces_idx.size(); ++i) {
        // First vertex
        faces[(i * 9)] = m.vertices[m.faces_idx[i][0]][0];
        faces[(i * 9) + 1] = m.vertices[m.faces_idx[i][0]][1];
        faces[(i * 9) + 2] = m.vertices[m.faces_idx[i][0]][2];


        // Second vertex
        faces[(i * 9) + 3] = m.vertices[m.faces_idx[i][1]][0];
        faces[(i * 9) + 4] = m.vertices[m.faces_idx[i][1]][1];
        faces[(i * 9) + 5] = m.vertices[m.faces_idx[i][1]][2];


        // Third vertex
        faces[(i * 9) + 6] = m.vertices[m.faces_idx[i][2]][0];
        faces[(i * 9) + 7] = m.vertices[m.faces_idx[i][2]][1];
        faces[(i * 9) + 8] = m.vertices[m.faces_idx[i][2]][2];
    }

    // Setup mesh faces inside GPU
    gpuErrchk(hipMalloc((void**)&faces_d, sizeof(float) * m.faces_idx.size() * 9));
    gpuErrchk(hipMemcpy(faces_d, faces, sizeof(float) * m.faces_idx.size() * 9, hipMemcpyHostToDevice));

    // Compute the grid and block dimensions
    int minGridSize;
    int blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, voxelize, 0, 0);
    dim3 grid((m.faces_idx.size() + blockSize - 1) / blockSize);

    gpuErrchk(hipEventRecord(start, 0));

    // Launch the voxelization kernel
    voxelize <<<grid, blockSize >>> (faces_d, m.faces_idx.size(), (size_t)v_grid.dim_x, (size_t)v_grid.dim_y, (size_t)v_grid.dim_z, v_table_d, make_float3(v_grid.spacing[0], v_grid.spacing[1], v_grid.spacing[2]), make_float3(v_grid.aabb.p_min[0], v_grid.aabb.p_min[1], v_grid.aabb.p_min[2]));
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipEventRecord(end, 0));
    gpuErrchk(hipEventSynchronize(end));
    gpuErrchk(hipEventElapsedTime(&elapsed_time, start, end));
    time = elapsed_time;

    // Copy the GPU memory to RAM
    gpuErrchk(hipMemcpy(v_table, v_table_d, sizeof(unsigned int) * (size_t)v_grid.dim_x * (size_t)v_grid.dim_y * (size_t)v_grid.dim_z, hipMemcpyDeviceToHost));
    
    // Free the GPU memory
    gpuErrchk(hipFree(v_table_d));
    gpuErrchk(hipFree(faces_d));
}